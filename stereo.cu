#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <sys/time.h>

#include <hipDNN.h>

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

#define ok(expr) if (expr != 0) { printf("ERROR on line %d\n", __LINE__); exit(-1); }

hipdnnHandle_t cudnn_handle;

/* Tensor */
struct Tensor {
	float *data;
	hipdnnTensorDescriptor_t desc;
	int n, c, h, w, size, capacity;
};

void Tensor_init(struct Tensor *t)
{
	t->data = NULL;
	t->n = t->c = t->h = t->w = t->size = t->capacity = 0;
	ok(hipdnnCreateTensorDescriptor(&t->desc));
}

void Tensor_resize(struct Tensor *t, int n, int c, int h, int w)
{
	int size = n * c * h * w;

	if (t->capacity != 0 && size > t->capacity) {
		printf("DNN: reallocating tensor\n");
		ok(hipFree(t->data));
		t->capacity = 0;
	}

	if (t->capacity == 0) {
		t->capacity = size;
		ok(hipMalloc(&t->data, t->capacity * 4));
	}
	t->n = n;
	t->c = c;
	t->h = h;
	t->w = w;
	t->size = size;

	ok(hipdnnSetTensor4dDescriptor(t->desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w))
}

void Tensor_init_resize(struct Tensor *t, int n, int c, int h, int w)
{
	Tensor_init(t);
	Tensor_resize(t, n, c, h, w);
}

void Tensor_print(struct Tensor *t)
{
	float *buf = (float *)malloc(t->size * 4);
	hipMemcpy(buf, t->data, t->size * 4, hipMemcpyDeviceToHost);
	printf("%d x %d x %d x %d\n", t->n, t->c, t->h, t->w);
	int i = 0;
	int j = 0;
	for (int k = 0; k < min(t->h, 6); k++) {
		for (int l = 0; l < min(t->w, 6); l++) {
			printf("%e ", buf[((i * t->c + j) * t->h + k) * t->w + l]);
		}
		printf("\n");
	}
	free(buf);
}


/* ConvLayer */
struct ConvLayer {
	int relu;
	struct Tensor output, weight, bias;
	hipdnnConvolutionFwdAlgo_t algorithm;
	hipdnnFilterDescriptor_t weight_desc;
	hipdnnConvolutionDescriptor_t conv_desc;
};

void ConvLayer_init(ConvLayer *e, int n_in, int n_out, int kw, int kh, int sx, int sy, int padw, int padh, int relu)
{
	Tensor_init(&e->output);
	Tensor_init_resize(&e->weight, n_out, n_in, kh, kw);
	Tensor_init_resize(&e->bias, 1, n_out, 1, 1);
	ok(hipdnnCreateFilterDescriptor(&e->weight_desc));
	ok(hipdnnSetFilter4dDescriptor(e->weight_desc, HIPDNN_DATA_FLOAT, n_out, n_in, kh, kw));
	ok(hipdnnCreateConvolutionDescriptor(&e->conv_desc));
	ok(hipdnnSetConvolution2dDescriptor(e->conv_desc, padh, padw, sy, sx, 1, 1, HIPDNN_CONVOLUTION));
	e->relu = relu;
}

struct Tensor *ConvLayer_allocate(ConvLayer *e, struct Tensor *i) {
	int n, c, h, w;
	ok(hipdnnGetConvolution2dForwardOutputDim(e->conv_desc, i->desc, e->weight_desc, &n, &c, &h, &w));
	Tensor_resize(&e->output, n, c, h, w);
	ok(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle, i->desc, e->weight_desc, e->conv_desc,
		e->output.desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &e->algorithm));
	return &e->output;
}

struct Tensor *ConvLayer_forward(ConvLayer *e, struct Tensor *i) {
	int zero = 0;
	int one = 1;
	ok(hipdnnConvolutionForward(cudnn_handle, &one, i->desc, i->data, e->weight_desc, e->weight.data,
		e->conv_desc, e->algorithm, NULL, 0, &zero, e->output.desc, e->output.data));
	ok(hipdnnAddTensor(cudnn_handle, CUDNN_ADD_SAME_C, &one, e->bias.desc, e->bias.data, &one,
		e->output.desc, e->output.data));
	if (e->relu) {
		ok(hipdnnActivationForward(cudnn_handle, HIPDNN_ACTIVATION_RELU, &one, e->output.desc, 
			e->output.data, &zero, e->output.desc, e->output.data));
	}
	return &e->output;
}

/* Sequential */
struct Sequential {
	struct ConvLayer modules[32];
	int num_modules;
};

void Sequential_load(struct Sequential *s, const char *dir)
{
	char desc_fname[32];

	snprintf(desc_fname, 32, "%s/desc", dir);
	FILE *f = fopen(desc_fname, "r");
	int n = fscanf(f, "%d\n", &s->num_modules);

	printf("load network from %s with %d conv layers\n", dir, s->num_modules);
	for (int i = 0; i < s->num_modules; i++) {
		int n_in, n_out, kw, kh, dw, dh, padw, padh, relu;
		n = fscanf(f, "%d %d %d %d %d %d %d %d %d\n", &n_in, &n_out, &kw, &kh, &dw, &dh, &padw, &padh, &relu);
		ConvLayer_init(s->modules + i, n_in, n_out, kw, kh, dw, dh, padw, padh, relu);
		printf("conv: %d %d %d %d %d %d %d %d %d\n", n_in, n_out, kw, kh, dw, dh, padw, padh, relu);
	}
}

struct Tensor *Sequential_allocate(struct Sequential *s, struct Tensor *input)
{
	struct Tensor *output = input;
	for (int i = 0; i < s->num_modules; i++) {
		output = ConvLayer_allocate(s->modules + i, output);
	}
	return output;
}

struct Tensor *Sequential_forward(struct Sequential *s, struct Tensor *input)
{
	struct Tensor *output = input;
	for (int i = 0; i < s->num_modules; i++) {
		output = ConvLayer_forward(s->modules + i, output);
	}
	return output;
}

double get_time()
{
	struct timeval t;
	struct timezone tzp;
	gettimeofday(&t, &tzp);
	return t.tv_sec + t.tv_usec * 1e-6;
}

__global__ void zero_(float *input, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = 0;
	}
}

void zero(Tensor *t)
{
	zero_<<<GS(t->size), TB>>>(t->data, t->size);
}

__global__ void add_(float *input, float value, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = input[id] + value;
	}
}

void add(Tensor *t, float value)
{
	add_<<<GS(t->size), TB>>>(t->data, value, t->size);
}

__global__ void mul_(float *input, float factor, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = input[id] * factor;
	}
}

void mul(Tensor *t, float factor)
{
	mul_<<<GS(t->size), TB>>>(t->data, factor, t->size);
}

void __global__ rgb2gray_(unsigned char *input, float *output, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		float b = input[id * 4 + 0];
		float g = input[id * 4 + 1];
		float r = input[id * 4 + 2];
		output[id] = 0.299 * r + 0.587 * g + 0.114 * b;
	}
}

void rgb2gray(unsigned char *input, Tensor *output, int h, int w)
{
	Tensor_resize(output, 1, 1, h, w);
	rgb2gray_<<<GS(output->size), TB>>>(input, output->data, output->size);
}

void __global__ gray2display_(float *input, unsigned char *display, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		display[id * 4 + 0] = input[id];
		display[id * 4 + 1] = input[id];
		display[id * 4 + 2] = input[id];
		display[id * 4 + 3] = 0;
	}
}

void gray2display(Tensor *input, unsigned char *display)
{
	gray2display_<<<GS(input->size), TB>>>(input->data, display, input->size);
}

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size023) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		float sum = 0.0;
		for (int dim1 = 0; dim1 < size1; dim1++) {
			float x = input[(dim0 * size1 + dim1) * size23 + dim23];
			sum += x * x;
		}
		norm[dim0 * size23 + dim23] = sum + 1.3e-37;
	}
}

__global__ void Normalize_forward_(float *input, float *norm, float *output, int size23, int size123, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) {
		int dim23 = id % size23;
		int dim0 = (id / size123);
		output[id] = input[id] / sqrtf(norm[dim0 * size23 + dim23]);
	}
}

void Normalize_forward(Tensor *input, Tensor *norm)
{
	Tensor_resize(norm, input->n, 1, input->h, input->w);
	Normalize_get_norm_<<<GS(norm->size), TB>>>(input->data, norm->data, input->c, 
		input->h * input->w, norm->size);
	Normalize_forward_<<<GS(input->size), TB>>>(input->data, norm->data, input->data,
		input->h * input->w, input->c * input->h * input->w, input->size);
}


__global__ void ad_(float *x0, float *x1, float *output, int size, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			int cnt = 0;
			dist = 0;
			for (int yy = y - 2; yy <= y + 2; yy++) {
				for (int xx = x - 2; xx <= x + 2; xx++) {
					if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
						int ind = yy * size3 + xx;
						dist += abs(x0[ind] - x1[ind + d]);
						cnt++;
					}
				}
			}
			dist /= cnt;
		} else {
			dist = HIP_NAN;
		}
		output[id] = dist;
	}
}

void ad(Tensor *x0, Tensor *x1, Tensor *mc, int disp_max, int direction)
{
	Tensor_resize(mc, 1, disp_max, x0->h, x0->w);
	ad_<<<GS(mc->size), TB>>>(x0->data, x1->data, mc->data, mc->size, mc->h, mc->w, direction);
}


__global__ void argmin_(float *input, float *output, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size23) {
		int argmin = 0;
		float min = HIP_INF;
		for (int i = 0; i < size1; i++) {
			float val = input[i * size23 + id];
			if (val < min) {
				min = val;
				argmin = i;
			}
		}
		output[id] = argmin;
	}
}

void argmin(Tensor *input, Tensor *output)
{
	Tensor_resize(output, 1, 1, input->h, input->w);
	argmin_<<<GS(output->size), TB>>>(input->data, output->data, input->c, output->size);
}


__global__ void downsample_(float *input, float *output, int factor, int size3, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim3 = id % size3;
		int dim2 = id / size3;
		atomicAdd(output + ((dim2 / factor) * (size3 / factor) + (dim3 / factor)), input[id] / (factor * factor));
	}
}

void downsample(Tensor *input, Tensor *output, int factor)
{
	assert(input->h % factor == 0);
	assert(input->w % factor == 0);
	Tensor_resize(output, 1, 1, input->h / factor, input->w / factor);
	zero(output);
	downsample_<<<GS(input->size), TB>>>(input->data, output->data, factor, input->w, input->size);
}

void load_batch(Tensor *x0, Tensor *x1, Tensor *batch)
{
	int size = x0->size * 4;
	Tensor_resize(batch, 2, 1, x0->h, x0->w);
	hipMemcpy(batch->data, x0->data, size, hipMemcpyDeviceToDevice);
	hipMemcpy(batch->data + size, x1->data, size, hipMemcpyDeviceToDevice);
}

Sequential net;
Tensor x0_gray_big, x1_gray_big, x0_gray, x1_gray, x0_mc, x0_disp, batch, norm;
int width_big, height_big, size_big, width, height, size;

int downsample_factor = 5;
int disp_max = 32;
const float mean = 95;
const float stddev = 65;

void stereo_init(int width_arg, int height_arg)
{
	Tensor_init(&x0_gray_big);
	Tensor_init(&x1_gray_big);
	Tensor_init(&x0_gray);
	Tensor_init(&x1_gray);
	Tensor_init(&x0_mc);
	Tensor_init(&x0_disp);
	Tensor_init(&batch);
	Tensor_init(&norm);

	width_big = width_arg;
	height_big = height_arg;
	size_big = width_big * height_big;

	assert(width_big % downsample_factor == 0);
	assert(height_big % downsample_factor == 0);

	width = width_big / downsample_factor;
	height = height_big / downsample_factor;
	size = width * height;

	ok(hipdnnCreate(&cudnn_handle));

	Tensor_resize(&batch, 2, 1, height, width);
	Sequential_load(&net, "tmp/foo");
	Sequential_allocate(&net, &batch);

	printf("stereo_init: %d x %d\n", width, height);
}

void stereo_run(unsigned char *x0, unsigned char *x1, unsigned char *display)
{
	rgb2gray(x0, &x0_gray_big, height_big, width_big);
	rgb2gray(x1, &x1_gray_big, height_big, width_big);

	downsample(&x0_gray_big, &x0_gray, downsample_factor);
	downsample(&x1_gray_big, &x1_gray, downsample_factor);

	// image preprocessing
	add(&x0_gray, -mean);
	mul(&x0_gray, 1 / stddev);
	add(&x1_gray, -mean);
	mul(&x1_gray, 1 / stddev);

	load_batch(&x0_gray, &x1_gray, &batch);
	Sequential_forward(&net, &batch);
	Normalize_forward(&batch, &norm);
	Tensor_print(&norm);

//	ad(&x0_gray, &x1_gray, &x0_mc, disp_max, -1);
//	argmin(&x0_mc, &x0_disp);

	// undo image preprocessing
	mul(&x0_gray, stddev);
	add(&x0_gray, mean);
	mul(&x1_gray, stddev);
	add(&x1_gray, mean);

	gray2display(&x0_gray, display);
	gray2display(&x1_gray, display + size * 4);
}
