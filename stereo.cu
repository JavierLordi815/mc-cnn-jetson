#include "hip/hip_runtime.h"
#include <assert.h>
#include <fcntl.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/types.h>

#include <hipDNN.h>

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

#define ok(expr) if (expr != 0) { printf("ERROR on line %d\n", __LINE__); exit(-1); }

hipdnnHandle_t cudnn_handle;

/* Tensor */
struct Tensor {
	float *data;
	hipdnnTensorDescriptor_t desc;
	int n, c, h, w, size, capacity;
};

void Tensor_init(struct Tensor *t)
{
	t->data = NULL;
	t->n = t->c = t->h = t->w = t->size = t->capacity = 0;
	ok(hipdnnCreateTensorDescriptor(&t->desc));
}

void Tensor_resize(struct Tensor *t, int n, int c, int h, int w)
{
	int size = n * c * h * w;

	if (t->capacity != 0 && size > t->capacity) {
		printf("DNN: reallocating tensor\n");
		ok(hipFree(t->data));
		t->capacity = 0;
	}

	if (t->capacity == 0) {
		t->capacity = size;
		ok(hipMalloc(&t->data, t->capacity * 4));
	}
	t->n = n;
	t->c = c;
	t->h = h;
	t->w = w;
	t->size = size;

	ok(hipdnnSetTensor4dDescriptor(t->desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w))
}

void Tensor_init_resize(struct Tensor *t, int n, int c, int h, int w)
{
	Tensor_init(t);
	Tensor_resize(t, n, c, h, w);
}

void Tensor_print_ext(struct Tensor *t, int n, int c, int h, int w)
{
	float *buf = (float *)malloc(t->size * 4);
	hipMemcpy(buf, t->data, t->size * 4, hipMemcpyDeviceToHost);
	printf("%d x %d x %d x %d; %d, %d, %d, %d\n", t->n, t->c, t->h, t->w, n, c, h, w);
	assert(n < t->n);
	assert(c < t->c);
	int i = n;
	int j = c;
	for (int k = h; k < min(t->h, h + 6); k++) {
		for (int l = w; l < min(t->w, w + 6); l++) {
			printf("%e ", buf[((i * t->c + j) * t->h + k) * t->w + l]);
		}
		printf("\n");
	}
	free(buf);
}

void Tensor_print(struct Tensor *t)
{
	Tensor_print_ext(t, 0, 0, 0, 0);
}

__global__ void fill_(float *input, float value, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = value;
	}
}

void fill(Tensor *t, float value)
{
	fill_<<<GS(t->size), TB>>>(t->data, value, t->size);
}

__global__ void add_(float *input, float value, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = input[id] + value;
	}
}

void add(Tensor *t, float value)
{
	add_<<<GS(t->size), TB>>>(t->data, value, t->size);
}

__global__ void mul_(float *input, float factor, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		input[id] = input[id] * factor;
	}
}

void mul(Tensor *t, float factor)
{
	mul_<<<GS(t->size), TB>>>(t->data, factor, t->size);
}

/* ConvLayer */
struct ConvLayer {
	int relu;
	struct Tensor output, weight, bias;
	hipdnnConvolutionFwdAlgo_t algorithm;
	hipdnnFilterDescriptor_t weight_desc;
	hipdnnConvolutionDescriptor_t conv_desc;
};

void ConvLayer_init(ConvLayer *e, int n_in, int n_out, int kw, int kh, int sx, int sy, int padw, int padh, int relu)
{
	Tensor_init(&e->output);
	Tensor_init_resize(&e->weight, n_out, n_in, kh, kw);
	Tensor_init_resize(&e->bias, 1, n_out, 1, 1);
	ok(hipdnnCreateFilterDescriptor(&e->weight_desc));
	ok(hipdnnSetFilter4dDescriptor(e->weight_desc, HIPDNN_DATA_FLOAT, n_out, n_in, kh, kw));
	ok(hipdnnCreateConvolutionDescriptor(&e->conv_desc));
	ok(hipdnnSetConvolution2dDescriptor(e->conv_desc, padh, padw, sy, sx, 1, 1, HIPDNN_CROSS_CORRELATION));
	e->relu = relu;
}

struct Tensor *ConvLayer_allocate(ConvLayer *e, struct Tensor *i) {
	int n, c, h, w;
	ok(hipdnnGetConvolution2dForwardOutputDim(e->conv_desc, i->desc, e->weight_desc, &n, &c, &h, &w));
	Tensor_resize(&e->output, n, c, h, w);
	fill(&e->output, 0);
	ok(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle, i->desc, e->weight_desc, e->conv_desc,
		e->output.desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &e->algorithm));
	return &e->output;
}

struct Tensor *ConvLayer_forward(ConvLayer *e, struct Tensor *i) {
	float zero = 0;
	float one = 1;
	ok(hipdnnConvolutionForward(cudnn_handle, &one, i->desc, i->data, e->weight_desc, e->weight.data,
		e->conv_desc, e->algorithm, NULL, 0, &zero, e->output.desc, e->output.data));
	ok(hipdnnAddTensor(cudnn_handle, CUDNN_ADD_SAME_C, &one, e->bias.desc, e->bias.data, &one,
		e->output.desc, e->output.data));
	if (e->relu) {
		ok(hipdnnActivationForward(cudnn_handle, HIPDNN_ACTIVATION_RELU, &one, e->output.desc, 
			e->output.data, &zero, e->output.desc, e->output.data));
	}
	return &e->output;
}

/* Sequential */
struct Sequential {
	struct ConvLayer modules[32];
	int num_modules;
};

void mmap2gpu(const char *fname, float *data, int size)
{
	int fd = open(fname, O_RDONLY);
	float *map = (float *)mmap(NULL, size * 4, PROT_READ, MAP_SHARED, fd, 0);
	hipMemcpy(data, map, size, hipMemcpyHostToDevice);
	close(fd);
}

void Sequential_load(struct Sequential *s, const char *dir)
{
	char buf[256];
	snprintf(buf, 256, "%s/desc", dir);
	FILE *f = fopen(buf, "r");
	int n = fscanf(f, "%d\n", &s->num_modules);

	printf("load network from %s with %d conv layers\n", dir, s->num_modules);
	for (int i = 0; i < s->num_modules; i++) {
		int n_in, n_out, kw, kh, dw, dh, padw, padh, relu;
		n = fscanf(f, "%d %d %d %d %d %d %d %d %d\n", &n_in, &n_out, &kw, &kh, &dw, &dh, &padw, &padh, &relu);
		ConvLayer_init(s->modules + i, n_in, n_out, kw, kh, dw, dh, padw, padh, relu);
		printf("conv: %d %d %d %d %d %d %d %d %d\n", n_in, n_out, kw, kh, dw, dh, padw, padh, relu);

		snprintf(buf, 256, "%s/%dW", dir, i);
		mmap2gpu(buf, s->modules[i].weight.data, s->modules[i].weight.size * 4);
		snprintf(buf, 256, "%s/%dB", dir, i);
		mmap2gpu(buf, s->modules[i].bias.data, s->modules[i].bias.size * 4);
	}
}

struct Tensor *Sequential_allocate(struct Sequential *s, struct Tensor *input)
{
	struct Tensor *output = input;
	for (int i = 0; i < s->num_modules; i++) {
		output = ConvLayer_allocate(s->modules + i, output);
	}
	return output;
}

struct Tensor *Sequential_forward(struct Sequential *s, struct Tensor *input)
{
	struct Tensor *output = input;
	for (int i = 0; i < s->num_modules; i++) {
		output = ConvLayer_forward(s->modules + i, output);
	}
	return output;
}

double get_time()
{
	struct timeval t;
	struct timezone tzp;
	gettimeofday(&t, &tzp);
	return t.tv_sec + t.tv_usec * 1e-6;
}

void __global__ rgb2gray_(unsigned char *input, float *output, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		float b = input[id * 4 + 0];
		float g = input[id * 4 + 1];
		float r = input[id * 4 + 2];
		output[id] = 0.299 * r + 0.587 * g + 0.114 * b;
	}
}

void rgb2gray(unsigned char *input, Tensor *output, int h, int w)
{
	Tensor_resize(output, 1, 1, h, w);
	rgb2gray_<<<GS(output->size), TB>>>(input, output->data, output->size);
}

void __global__ gray2display_(float *input, unsigned char *display, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		display[id * 4 + 0] = input[id];
		display[id * 4 + 1] = input[id];
		display[id * 4 + 2] = input[id];
		display[id * 4 + 3] = 0;
	}
}

void gray2display(Tensor *input, unsigned char *display)
{
	gray2display_<<<GS(input->size), TB>>>(input->data, display, input->size);
}

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size023) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		float sum = 0.0;
		for (int dim1 = 0; dim1 < size1; dim1++) {
			float x = input[(dim0 * size1 + dim1) * size23 + dim23];
			sum += x * x;
		}
		norm[dim0 * size23 + dim23] = sum + 1.3e-37;
	}
}

__global__ void Normalize_forward_(float *input, float *norm, float *output, int size23, int size123, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) {
		int dim23 = id % size23;
		int dim0 = (id / size123);
		output[id] = input[id] / sqrtf(norm[dim0 * size23 + dim23]);
	}
}

void Normalize_forward(Tensor *input, Tensor *norm)
{
	Tensor_resize(norm, input->n, 1, input->h, input->w);
	Normalize_get_norm_<<<GS(norm->size), TB>>>(input->data, norm->data, input->c, 
		input->h * input->w, norm->size);
	Normalize_forward_<<<GS(input->size), TB>>>(input->data, norm->data, input->data,
		input->h * input->w, input->c * input->h * input->w, input->size);
}

__global__ void StereoJoin_(float *input_L, float *input_R, float *output_L, float *output_R, int size1_input, int size1, int size3, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size23) {
		int dim3 = id % size3;
		assert(size1_input <= 32);
		float L_cache[32];
		for (int i = 0; i < size1_input; i++) {
			L_cache[i] = input_L[i * size23 + id];
		}

		for (int d = 0; d < size1; d++) {
			if (dim3 - d >= 0) {
				float sum = 0;
				for (int i = 0; i < size1_input; i++) {
					sum -= L_cache[i] * input_R[i * size23 + id - d];
				}
				output_L[d * size23 + id] = sum;
				output_R[d * size23 + id - d] = sum;
			}
		}
	}
}

void StereoJoin(Tensor *input, Tensor *output_L, Tensor *output_R, int disp_max)
{
	Tensor_resize(output_L, 1, disp_max, input->h, input->w);
	Tensor_resize(output_R, 1, disp_max, input->h, input->w);
	fill(output_L, NAN);
	fill(output_R, NAN);
	int size23 = input->h * input->w;
	StereoJoin_<<<GS(size23), TB>>>(input->data, input->data + input->size / 2, output_L->data, output_R->data,
		input->c, output_L->c, output_L->w, size23);
}

__global__ void ad_(float *x0, float *x1, float *output, int size, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			int cnt = 0;
			dist = 0;
			for (int yy = y - 2; yy <= y + 2; yy++) {
				for (int xx = x - 2; xx <= x + 2; xx++) {
					if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
						int ind = yy * size3 + xx;
						dist += abs(x0[ind] - x1[ind + d]);
						cnt++;
					}
				}
			}
			dist /= cnt;
		} else {
			dist = HIP_NAN_F;
		}
		output[id] = dist;
	}
}

void ad(Tensor *x0, Tensor *x1, Tensor *mc, int disp_max, int direction)
{
	Tensor_resize(mc, 1, disp_max, x0->h, x0->w);
	ad_<<<GS(mc->size), TB>>>(x0->data, x1->data, mc->data, mc->size, mc->h, mc->w, direction);
}


__global__ void argmin_(float *input, float *output, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size23) {
		int argmin = 0;
		float min = HIP_INF_F;
		for (int i = 0; i < size1; i++) {
			float val = input[i * size23 + id];
			if (val < min) {
				min = val;
				argmin = i;
			}
		}
		output[id] = argmin;
	}
}

void argmin(Tensor *input, Tensor *output)
{
	Tensor_resize(output, 1, 1, input->h, input->w);
	argmin_<<<GS(output->size), TB>>>(input->data, output->data, input->c, output->size);
}


__global__ void downsample_(float *input, float *output, int factor, int size3, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim3 = id % size3;
		int dim2 = id / size3;
		atomicAdd(output + ((dim2 / factor) * (size3 / factor) + (dim3 / factor)), input[id] / (factor * factor));
	}
}

void downsample(Tensor *input, Tensor *output, int factor)
{
	assert(input->h % factor == 0);
	assert(input->w % factor == 0);
	Tensor_resize(output, 1, 1, input->h / factor, input->w / factor);
	fill(output, 0);
	downsample_<<<GS(input->size), TB>>>(input->data, output->data, factor, input->w, input->size);
}

void load_batch(Tensor *x0, Tensor *x1, Tensor *batch)
{
	Tensor_resize(batch, 2, 1, x0->h, x0->w);
	hipMemcpy(batch->data, x0->data, x0->size * 4, hipMemcpyDeviceToDevice);
	hipMemcpy(batch->data + x0->size, x1->data, x0->size * 4, hipMemcpyDeviceToDevice);
}

Sequential net;
Tensor x0_gray_big, x1_gray_big, x0_gray, x1_gray, x0_mc, x1_mc, x0_disp, x1_disp, batch, norm;
int width_big, height_big, size_big, width, height, size;

int downsample_factor = 10;
int disp_max = 32;
const float mean = 95;
const float stddev = 65;

void stereo_init(int width_arg, int height_arg)
{
	Tensor_init(&x0_gray_big);
	Tensor_init(&x1_gray_big);
	Tensor_init(&x0_gray);
	Tensor_init(&x1_gray);
	Tensor_init(&x0_mc);
	Tensor_init(&x1_mc);
	Tensor_init(&x0_disp);
	Tensor_init(&x1_disp);
	Tensor_init(&batch);
	Tensor_init(&norm);

	width_big = width_arg;
	height_big = height_arg;
	size_big = width_big * height_big;

	assert(width_big % downsample_factor == 0);
	assert(height_big % downsample_factor == 0);

	width = width_big / downsample_factor;
	height = height_big / downsample_factor;
	size = width * height;

	ok(hipdnnCreate(&cudnn_handle));

	Tensor_resize(&batch, 2, 1, height, width);
	Sequential_load(&net, "net/net_kitti_fast_-a_train_tr_-fm_16");
	Sequential_allocate(&net, &batch);

	printf("stereo_init: %d x %d\n", width, height);
}

void stereo_run(unsigned char *x0, unsigned char *x1, unsigned char *display)
{
	rgb2gray(x0, &x0_gray_big, height_big, width_big);
	rgb2gray(x1, &x1_gray_big, height_big, width_big);

	downsample(&x0_gray_big, &x0_gray, downsample_factor);
	downsample(&x1_gray_big, &x1_gray, downsample_factor);

	// image preprocessing
	add(&x0_gray, -mean);
	mul(&x0_gray, 1 / stddev);
	add(&x1_gray, -mean);
	mul(&x1_gray, 1 / stddev);

	// network
	load_batch(&x0_gray, &x1_gray, &batch);
	Tensor *output = Sequential_forward(&net, &batch);
	Normalize_forward(output, &norm);
	StereoJoin(output, &x0_mc, &x1_mc, disp_max);

	// stereo method
	argmin(&x0_mc, &x0_disp);
	mul(&x0_disp, 8);

	// absolute differences
	// ad(&x0_gray, &x1_gray, &x0_mc, disp_max, -1);
	// argmin(&x0_mc, &x0_disp);

	// undo image preprocessing
	mul(&x0_gray, stddev);
	add(&x0_gray, mean);
	mul(&x1_gray, stddev);
	add(&x1_gray, mean);

	gray2display(&x0_gray, display);
	gray2display(&x0_disp, display + size * 4);
}
